
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <omp.h>

// // Number of steps to take between the lower and upper bound
// #define N 10000000

__global__ void initCalcKernel(double *x_dev, double *y_dev, double x1, double stepSize, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // Ensure that i does not exceed N
    if(i < N) {
        // Calculate each value of x used to calculate f(x)
        x_dev[i] = x1 + (i * stepSize);
        __syncthreads();
        y_dev[i] = exp(-((x_dev[i]-2) * (x_dev[i]-2))) + exp(-((x_dev[i]-6) * (x_dev[i]-6)) / 10) + (1 / ((x_dev[i] * x_dev[i]) + 1));
    }
}

int main(int argc, char *argv[]) 
{
    omp_set_num_threads(16);
    int i;
    char *ptr;
    int N = strtol(argv[1], &ptr, 10);

    // Set up timers
    hipEvent_t initCalcStart, initCalcStop, memcpyStart, memcpyStop;
    hipEventCreate(&initCalcStart); hipEventCreate(&initCalcStop);
    hipEventCreate(&memcpyStart); hipEventCreate(&memcpyStop);
    double maximisationStart, maximisationStop;
    double totalStart, totalStop;
    
    totalStart = omp_get_wtime();

    // Set up the problem
    double x1 = -100.0, x2 = 100.0; // Upper and lower bounds of x
    double stepSize = (x2 - x1) / N; // Size of the steps between the upper and lower bound

    // Calculate the number of blocks and threads per block to use
    int threadsPerBlock, maxThreadsPerBlock=1024;
    threadsPerBlock = N > maxThreadsPerBlock ? maxThreadsPerBlock : N;
    int blocks = ceil(N / threadsPerBlock);

    // Define variable to hold values of x and y on host and GPU
    double *x, *x_dev, *y, *y_dev;
    x = (double *) malloc(N*sizeof(*x));
    hipMalloc(&x_dev, N*sizeof(double));
    y = (double *) malloc(N*sizeof(*y));
    hipMalloc(&y_dev, N*sizeof(double));

    // Call initialisation kernel
    hipEventRecord(initCalcStart,0);
    initCalcKernel<<<blocks, threadsPerBlock>>> (x_dev, y_dev, x1, stepSize, N);
    hipEventRecord(initCalcStop,0);

    // Check for errors
    hipError_t e = hipGetLastError();
    if(e != hipSuccess) {
      printf("CUDA ERROR: %s \n", hipGetErrorString(e));
    }

    // Bring data back to host
    hipEventRecord(memcpyStart,0);
    hipMemcpy(x, x_dev, N*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(y, y_dev, N*sizeof(double), hipMemcpyDeviceToHost);
    hipEventRecord(memcpyStop,0);

    // Check for errors
    e = hipGetLastError();
    if(e != hipSuccess) {
      printf("CUDA ERROR: %s \n", hipGetErrorString(e));
    }

    // Syncrhonise host and GPU
    hipDeviceSynchronize();

    // Variables to store the maximum value of f(x)
    // and the value of x that maximises f(x)
    double max = y[0]; 
    double my_max = y[0];
    double max_x = 0;
    double my_max_x = 0; 

    // Calculate the maximum of f(x)
    maximisationStart = omp_get_wtime();
    #pragma omp parallel default(none) shared(x, y, max, max_x, N) private(i, my_max, my_max_x)
    {
        #pragma omp for
        for(i = 0; i < N; i++) {
            if(y[i] > my_max) {
                my_max = y[i];
                my_max_x = x[i];
            }
        }

        #pragma omp critical
        {
            if(my_max > max) {
                max = my_max;
                max_x = my_max_x;
            }
        }
    }
    maximisationStop = omp_get_wtime();
    totalStop = omp_get_wtime();

    // CUDA timing
    float initCalcTime, memcpyTime;
    hipEventElapsedTime(&initCalcTime, initCalcStart, initCalcStop);
    hipEventElapsedTime(&memcpyTime, memcpyStart, memcpyStop);

    // Print output to command line - we divide CUDA's timings by 1000 to convert from milliseconds to seconds
    printf("----------------------CUDA IMPLEMENTATION OUTPUT----------------------\n");
    printf("Matthew Carter | Student ID 201371920 | COMP528 Assignment 3 Output\n");
    printf("--------------------------------OUTPUT--------------------------------\n");
    printf("Calcuating maximum with %d steps and a step size of %.8f\n", N, stepSize);
    printf("Using %d blocks and %d threads per block\n", blocks, threadsPerBlock);
    printf("--------------------------------RESULT--------------------------------\n");
    printf("Maximum value of f(x) = %.15f when x = %.15f\n", max, max_x);
    printf("--------------------------------TIMING--------------------------------\n");
    printf("Time taken to initialise and calculate f(x) %f seconds\n", (initCalcTime / 1000));
    printf("Time taken to copy data to host %f seconds\n", (memcpyTime / 1000));
    printf("Time taken to maximse f(x) %f seconds\n", (maximisationStop - maximisationStart));
    printf("Total time taken %f seconds\n", (totalStop - totalStart));
    printf("----------------------------------------------------------------------\n");


    // Free allocated memory
    free(x);
    free(y);
    hipFree(x_dev);
    hipFree(y_dev);

    return 0;

}