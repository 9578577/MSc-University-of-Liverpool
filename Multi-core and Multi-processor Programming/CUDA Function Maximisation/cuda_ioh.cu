
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <omp.h>

__global__ void functionKernel(double *x_dev, double *y_dev, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // Ensure that i does not exceed N
    if(i < N) {
        // Calculate the value of f(x)
        y_dev[i] = exp(-((x_dev[i]-2) * (x_dev[i]-2))) + exp(-((x_dev[i]-6) * (x_dev[i]-6)) / 10) + (1 / ((x_dev[i] * x_dev[i]) + 1));
    }
}

int main(int argc, char *argv[]) 
{
    omp_set_num_threads(16); // We will always use 16 threads
    int i;
    char *ptr;
    // Number of iterations to take between the upper and lower bound
    // If no value is specified, default to 30 million
    int N = strtol(argv[1], &ptr, 10);

    // Set up timers
    hipEvent_t xCopyStart, xCopyStop, yCopyStart, yCopyStop, functionStart, functionStop;
    double initialisationStart, initialisationStop;
    hipEventCreate(&xCopyStart); hipEventCreate(&xCopyStop);
    hipEventCreate(&yCopyStart); hipEventCreate(&yCopyStop);
    hipEventCreate(&functionStart); hipEventCreate(&functionStop);
    double maximisationStart, maximisationStop;
    double totalStart, totalStop;

    totalStart = omp_get_wtime();
    // Set up the problem
    double x1 = -100.0, x2 = 100.0; // Upper and lower bounds of x
    double stepSize = (x2 - x1) / N; // Size of the steps between the upper and lower bound

    // Calculate the number of blocks and threads per block to use
    int threadsPerBlock, maxThreadsPerBlock=1024;
    threadsPerBlock = N > maxThreadsPerBlock ? maxThreadsPerBlock : N;
    int blocks = ceil(N / threadsPerBlock);

    // Define variable to hold values of x and y on host and GPU
    double *x, *x_dev, *y, *y_dev;
    x = (double *) malloc(N*sizeof(*x));
    hipMalloc(&x_dev, N*sizeof(double));
    y = (double *) malloc(N*sizeof(*y));
    hipMalloc(&y_dev, N*sizeof(double));

    // Populate the array "x" with values of x
    initialisationStart = omp_get_wtime();
    #pragma omp parallel for default(none) shared(x, stepSize, x1, N) \
    private(i) schedule(guided)
    for(i = 0; i < N; i++) {
        x[i] = x1 + (i * stepSize);
    }
    initialisationStop = omp_get_wtime();

    // Copy x from host to device
    hipEventRecord(xCopyStart, 0);
    hipMemcpy(x_dev, x, N*sizeof(double), hipMemcpyHostToDevice);
    hipEventRecord(xCopyStop,0);

    // Check for errors
    hipError_t e = hipGetLastError();
    if(e != hipSuccess) {
      printf("CUDA ERROR: %s \n", hipGetErrorString(e));
    }

    // Call function kernel
    hipEventRecord(functionStart,0);
    functionKernel<<<blocks, threadsPerBlock>>> (x_dev, y_dev, N);
    hipEventRecord(functionStop,0);

    // Check for errors
    e = hipGetLastError();
    if(e != hipSuccess) {
      printf("CUDA ERROR: %s \n", hipGetErrorString(e));
    }

    // Copy y from device to host
    hipEventRecord(yCopyStart, 0);
    hipMemcpy(y, y_dev, N*sizeof(double), hipMemcpyDeviceToHost);
    hipEventRecord(yCopyStop,0);

    // Check for errors
    e = hipGetLastError();
    if(e != hipSuccess) {
      printf("CUDA ERROR: %s \n", hipGetErrorString(e));
    }

    // Syncrhonise host and GPU
    hipDeviceSynchronize();

    // Variables to store the maximum value of f(x)
    // and the value of x that maximises f(x)
    double max = y[0]; 
    double my_max = y[0];
    double max_x = 0;
    double my_max_x = 0; 

    // Calculate the maximum of f(x)
    maximisationStart = omp_get_wtime();
    #pragma omp parallel default(none) shared(x, y, max, max_x, N) private(i, my_max, my_max_x)
    {
        // Calculate the maximum value of f(x) for each thread
        #pragma omp for
        for(i = 0; i < N; i++) {
            if(y[i] > my_max) {
                my_max = y[i];
                my_max_x = x[i];
            }
        }
        // Calculate the global maximum value of f(x)
        #pragma omp critical
        {
            if(my_max > max) {
                max = my_max;
                max_x = my_max_x;
            }
        }
    }
    maximisationStop = omp_get_wtime();
    totalStop = omp_get_wtime();

    // CUDA timing
    float functionTime, xCopyTime, yCopyTime;
    hipEventElapsedTime(&xCopyTime, xCopyStart, xCopyStop);
    hipEventElapsedTime(&functionTime, functionStart, functionStop);
    hipEventElapsedTime(&yCopyTime, yCopyStart, yCopyStop);

    // Print output to command line - we divide CUDA's timings by 1000 to convert from milliseconds to seconds
    printf("----------------------CUDA IMPLEMENTATION OUTPUT----------------------\n");
    printf("Matthew Carter | Student ID 201371920 | COMP528 Assignment 3 Output\n");
    printf("--------------------------------OUTPUT--------------------------------\n");
    printf("Calcuating maximum with %d steps and a step size of %.8f\n", N, stepSize);
    printf("Using %d blocks and %d threads per block\n", blocks, threadsPerBlock);
    printf("--------------------------------RESULT--------------------------------\n");
    printf("Maximum value of f(x) = %.15f when x = %.15f\n", max, max_x);
    printf("--------------------------------TIMING--------------------------------\n");
    printf("Time taken to initialise the problem %f seconds\n", (initialisationStop - initialisationStart));
    printf("Time taken to copy data to device %f seconds\n", (xCopyTime / 1000));
    printf("Time taken to calculate f(x) %f seconds\n", (functionTime / 1000));
    printf("Time taken to copy data to host %f seconds\n", (yCopyTime / 1000));
    printf("Time taken to maximse f(x) %f seconds\n", (maximisationStop - maximisationStart));
    printf("Total time taken %f seconds\n", (totalStop - totalStart));
    printf("----------------------------------------------------------------------\n");


    // Free allocated memory
    free(x);
    free(y);
    hipFree(x_dev);
    hipFree(y_dev);

    return 0;

}